#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "dfft_cuda.cuh"

// redistribute between group-cyclic distributions with different cycles
// (direction from block to cyclic)
__global__ void gpu_b2c_pack_kernel(unsigned int local_size,
                                    unsigned int ratio,
                                    unsigned int size,
                                    unsigned int npackets,
                                    unsigned int stride,
                                    cuda_cpx_t *local_data,
                                    cuda_cpx_t *send_data
                                    )
    {
    // index of local component
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // do not read beyond end of array
    if (idx >= local_size) return;

    unsigned int j = (idx/stride) % npackets; // packet number
    unsigned int r = (idx/stride - j)/ratio; // index in packet

    unsigned int offset = j*size;
    send_data[offset + r*stride + (idx%stride)] = local_data[idx];
    }

void gpu_b2c_pack(unsigned int local_size,
                  unsigned int ratio,
                  unsigned int size,
                  unsigned int npackets,
                  unsigned int stride,
                  cuda_cpx_t *local_data,
                  cuda_cpx_t *send_data)
    {
    unsigned int block_size =512;
    unsigned int n_blocks = local_size/block_size;
    if (local_size % block_size) n_blocks++;

    gpu_b2c_pack_kernel<<<n_blocks, block_size>>>(local_size,
                                                  ratio,
                                                  size,
                                                  npackets,
                                                  stride,
                                                  local_data,
                                                  send_data);
    }

// apply twiddle factors
__global__ void gpu_twiddle_kernel(unsigned int local_size,
                                   const unsigned int length,
                                   const unsigned int stride,
                                   float alpha,
                                   cuda_cpx_t *d_in,
                                   cuda_cpx_t *d_out,
                                   int inv)
    {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= local_size) return;

    int j = idx/stride;
    if (j >= length) return;
    float theta = -2.0f * float(M_PI) * alpha/(float) length;
    cuda_cpx_t w;
    CUDA_RE(w) = cosf((float)j*theta);
    CUDA_IM(w) = sinf((float)j*theta);

    cuda_cpx_t in = d_in[idx];
    cuda_cpx_t out;
    float sign = inv ? -1.0f : 1.0f;

    w.y *= sign;

    CUDA_RE(out) = CUDA_RE(in) * CUDA_RE(w) - CUDA_IM(in) * CUDA_IM(w);
    CUDA_IM(out) = CUDA_RE(in) * CUDA_IM(w) + CUDA_IM(in) * CUDA_RE(w); 

    d_out[idx] = out;
    }

void gpu_twiddle(unsigned int local_size,
                 const unsigned int length,
                 const unsigned int stride,
                 float alpha,
                 cuda_cpx_t *d_in,
                 cuda_cpx_t *d_out,
                 int inv)
    {
    unsigned int block_size =512;
    unsigned int n_block = local_size/block_size;
    if (local_size % block_size ) n_block++;

    gpu_twiddle_kernel<<<n_block, block_size>>>(local_size,
                                                length,
                                                stride,
                                                alpha,
                                                d_in,
                                                d_out,
                                                inv);
}

__global__ void gpu_c2b_unpack_kernel(const unsigned int local_size,
                                      const unsigned int length,
                                      const unsigned int c0,
                                      const unsigned int c1, 
                                      const unsigned int size,
                                      const unsigned int j0,
                                      const unsigned int stride,
                                      int rev,
                                      cuda_cpx_t *d_local_data,
                                      const cuda_cpx_t *d_scratch)
    {
    unsigned int idx = blockDim.x*blockIdx.x+threadIdx.x;

    if (idx >= local_size) return;

    // source processor
    int r = idx/size; // packet index
    int j1, j1_offset, del;
    int j0_remote = j0 + r*c1;
    if (rev && (length >= c0))
        {
        j1_offset = j0_remote*length/c0;
        del = 1;
        }
    else
        {
        j1_offset = j0_remote/c1;
        del = c0/c1;
        }

    // local index
    j1 = j1_offset + ((idx%size)/stride)*del;
    
    d_local_data[j1*stride+idx%stride] = d_scratch[idx];
    }

void gpu_c2b_unpack(const unsigned int local_size,
                    const unsigned int length,
                    const unsigned int c0,
                    const unsigned int c1, 
                    const unsigned int size,
                    const unsigned int j0,
                    const unsigned int stride,
                    const int rev,
                    cuda_cpx_t *d_local_data,
                    const cuda_cpx_t *d_scratch)
    {
    unsigned int block_size =512;
    unsigned int n_block = local_size/block_size;
    if (local_size % block_size ) n_block++;

    gpu_c2b_unpack_kernel<<<n_block, block_size>>>(local_size,
                                                   length,
                                                   c0,
                                                   c1,
                                                   size,
                                                   j0,
                                                   stride,
                                                   rev,
                                                   d_local_data,
                                                   d_scratch);
    }

__global__ void gpu_transpose_kernel(const unsigned int size,
                                     const unsigned int length,
                                     const unsigned int stride,
                                     const unsigned int embed,
                                     const cuda_cpx_t *in,
                                     cuda_cpx_t *out)
    {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if (idx >= size) return;

    int i = idx / stride;
    if (i >= length) return;

    int j = idx % stride;

    out[j*embed + i] = in[idx];
    }

void gpu_transpose(const unsigned int size,
                   const unsigned int length,
                   const unsigned int stride,
                   const unsigned int embed,
                   const cuda_cpx_t *in,
                   cuda_cpx_t *out)
    {
    unsigned int block_size =512;
    unsigned int n_block = size/block_size;
    if (size % block_size ) n_block++;
    
    gpu_transpose_kernel<<<n_block, block_size>>>(size, length, stride, embed, in, out);
    }
